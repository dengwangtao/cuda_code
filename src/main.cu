
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void say_hello(void)
{
    printf("Hello, world!\n");
}


int main(void)
{

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);

    say_hello <<<1,10>>>();
    hipDeviceReset();

    printf("Hello, world! Done.!\n");
    return 0;
}