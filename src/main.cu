#include <stdio.h>
#include <vector>
#include "common.cuh"
#include "add.cuh"
#include "cuda_utils.cuh"

int main(void)
{
    sz n = 10000;

    std::vector<int> vec1;
    std::vector<int> vec2;
    std::vector<int> result(n);

    for (sz i = 0; i < n; i++)
    {
        vec1.push_back(CommonUtils::random(0, 100));
        vec2.push_back(CommonUtils::random(0, 100));
    }

    // cuda add

    {
        auto d_vec1 = CudaMemRAII<int>::Alloc(n);
        auto d_vec2 = CudaMemRAII<int>::Alloc(n);

        d_vec1.CopyFromHost(vec1.data(), n);
        d_vec2.CopyFromHost(vec2.data(), n);

        auto d_result = CudaMemRAII<int>::Alloc(n);

        add <<<(n + 1023) / 1024, 1024>>> (d_vec1.get(), d_vec2.get(), d_result.get(), n);
        hipDeviceSynchronize();

        d_result.CopyToHost(result.data(), n);
    }

    for (sz i = 0; i < n; i++)
    {
        printf("%d + %d = %d\n", vec1[i], vec2[i], result[i]);
    }

    hipDeviceReset();
    return 0;
}